#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectorArithmetic(float *z, const float *x, const float *y, float A, int num) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Total number of threads
    for (int i = index; i < num; i += stride) {
        z[i] = A * x[i] + y[i];
    }
}

int main(void) {
    const int num = 1000000;  // Increase num to 1,000,000
    float *z, *x, *y;
    float *d_z, *d_x, *d_y;
    float A = 34;

    z = (float*) malloc(num * sizeof(float));
    x = (float*) malloc(num * sizeof(float));
    y = (float*) malloc(num * sizeof(float));

    for(int i = 0; i < num; i++) {
        x[i] = i;
        y[i] = 7 * i;
    }

    hipMalloc((void**)&d_z, num * sizeof(float));
    hipMalloc((void**)&d_x, num * sizeof(float));
    hipMalloc((void**)&d_y, num * sizeof(float));

    hipMemcpy(d_x, x, num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, num * sizeof(float), hipMemcpyHostToDevice);

    // Timing the kernel execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Experiment with these values
    int blockSize = 256;
    int gridSize = (num + blockSize - 1) / blockSize;
    vectorArithmetic<<<gridSize, blockSize>>>(d_z, d_x, d_y, A, num);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(z, d_z, num * sizeof(float), hipMemcpyDeviceToHost);

    // Output timing result
    printf("Time taken: %f ms\n", milliseconds);

    hipFree(d_z);
    hipFree(d_x);
    hipFree(d_y);

    free(z);
    free(x);
    free(y);

    return 0;
}
